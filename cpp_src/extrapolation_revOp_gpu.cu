#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include <iostream>
#include "stdio.h"
#include "wfPad.h"
#include <cmath>
#include <vector>
#include "revOp.h"

#include <chrono>
#include <ctime>

extern "C"
{

/*
-----------------------------------------------------------------
*/
__global__ void copyPadded(fcomp * paste, fcomp * copyied, \
    int nf, int nx, int M)
{
    int dim_x = nx+2*M;
    int pixelIdx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int pixelIdx_y = blockIdx.y * blockDim.y + threadIdx.y;

    if(pixelIdx_x < nx && pixelIdx_y < nf){
        int pixelIdx = pixelIdx_y * dim_x + pixelIdx_x + M;
        paste[pixelIdx] = copyied[pixelIdx];
    }
}

/*
------------------------------------------------------------------
*/
__global__ void imaging(fcomp * image, fcomp * forw_pulse, fcomp * back_pulse, \
    int nf, int nx, int M)
{
    int dim_x = nx+2*M;
    int pixelIdx_x = blockIdx.x * blockDim.x + threadIdx.x;

    fcomp conv;

    for(int j=0; j<nf; j++){
        int Idx = j * dim_x + pixelIdx_x + M;
        conv += forw_pulse[Idx] * thrust::conj(back_pulse[Idx]);
    }

    image[pixelIdx_x] = conv;
}

/*
------------------------------------------------------------------
*/
__global__ void extrapDepth(fcomp * new_wf, int nf, int nx, \
    int M, fcomp * w_op, fcomp * old_wf)
{
    int dim_x = nx+2*M;
    int length_M = 2*M+1;

    int xIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int fIdx = blockIdx.y * blockDim.y + threadIdx.y;

    fcomp pixel = fcomp(0.0,0.0);

    if(xIdx < nx && fIdx < nf){

        for(int k=0; k<length_M; ++k){
            pixel += w_op[fIdx*nx*length_M + k*nx + xIdx] * \
                old_wf[fIdx*dim_x + xIdx + k];
        }

        new_wf[fIdx*dim_x + M + xIdx] = pixel;
    }

} // end extrapolation to next depth


/*
------------------------------------------------------------------
*/
void extrapolate(int ns, int nextrap, int nz, int nt, int nf, int nx, int M,\
    fcomp * w_op_forw, fcomp * forw_pulse, fcomp * w_op_back, fcomp * back_pulse,\
    float * image)
{
    //define important dimensionality parameters
    int length_M = 2*M+1;
    int dim_x = nx+2*M;

    size_t sizePulse = nf * dim_x;
    size_t sizeAllSources = ns * sizePulse;
    size_t sizeOp = nextrap * nf * nx * length_M;
    size_t sizeImage = nz * nx;
    size_t sizeAllImages = ns * sizeImage;

    //rearrange operators
    auto startTime = std::chrono::high_resolution_clock::now();
    fcomp * h_w_op_forw = reverseOperator(w_op_forw, nextrap, nf, nx, length_M); //reverse operator's last two indices on host
    fcomp * h_w_op_back = reverseOperator(w_op_back, nextrap, nf, nx, length_M); //reverse operator's last two indices on host
    auto stopTime = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsedTime = stopTime - startTime;
    std::cout << "Reverse operators time(s): " << elapsedTime.count() / 1000.0 << " seconds." << std::endl;
    
    //allocate device memory
    fcomp * d_image;
    hipMalloc(&d_image, sizeAllImages * sizeof(fcomp));

    fcomp * d_w_op_forw, * d_old_forw, * d_new_forw;
    hipMalloc(&d_w_op_forw, sizeOp * sizeof(fcomp));
    hipMalloc(&d_old_forw, sizeAllSources * sizeof(fcomp));
    hipMalloc(&d_new_forw, sizeAllSources * sizeof(fcomp));

    fcomp * d_w_op_back, * d_old_back, * d_new_back;
    hipMalloc(&d_w_op_back, sizeOp * sizeof(fcomp));
    hipMalloc(&d_old_back, sizeAllSources * sizeof(fcomp));
    hipMalloc(&d_new_back, sizeAllSources * sizeof(fcomp));
    
    //copy operators on device
    hipMemcpy(d_w_op_forw, h_w_op_forw, sizeOp*sizeof(fcomp), hipMemcpyHostToDevice);
    hipMemcpy(d_w_op_back, h_w_op_back, sizeOp*sizeof(fcomp), hipMemcpyHostToDevice);

    startTime = std::chrono::high_resolution_clock::now();
    //allocate and read wavefields
    fcomp * h_image = new fcomp[sizeAllImages];
    std::vector<wfpad> h_forw_pulses(ns);
    std::vector<wfpad> h_back_pulses(ns);
    for(int is=0; is<ns; ++is){
        h_forw_pulses[is] = wfpad(nf, nx, 1, M, 0, &forw_pulse[is*nt*nx]);
        h_back_pulses[is] = wfpad(nf, nx, 1, M, 0, &back_pulse[is*nt*nx]);
    }
    stopTime = std::chrono::high_resolution_clock::now();
    elapsedTime = stopTime - startTime;
    std::cout << "Construct padded wavefields time(s): " << elapsedTime.count() / 1000.0 << " seconds." << std::endl;

    //define number of blocks and number of threads per block
    //define number of blocks and number of threads per block
    dim3 nThreads(32, 1, 1);
    size_t nBlocks_x = nx % nThreads.x == 0 ? size_t(nx/nThreads.x) : size_t(1 + nx/nThreads.x);
    size_t nBlocks_y = nf % nThreads.y == 0 ? size_t(nf/nThreads.y) : size_t(1 + nf/nThreads.y);
    size_t nBlocks_z = 1;
    dim3 nBlocks(nBlocks_x, nBlocks_y, nBlocks_z);
    std::cout << "nThreads: (" << nThreads.x << ", " << nThreads.y << ", " << nThreads.z << ")" << std::endl;
    std::cout << "nBlocks: (" << nBlocks.x << ", " << nBlocks.y << ", " << nBlocks.z << ")" << std::endl;

    //create one stream per source
    hipStream_t streams[ns];

    for(int is=0; is<ns; ++is){

        hipStreamCreate(&streams[is]);

        hipMemcpyAsync(&d_old_forw[is*sizePulse], h_forw_pulses[is].wf, \
            sizePulse*sizeof(fcomp), hipMemcpyHostToDevice, streams[is]);
        hipMemcpyAsync(&d_old_back[is*sizePulse], h_back_pulses[is].wf, \
            sizePulse*sizeof(fcomp), hipMemcpyHostToDevice, streams[is]);

        for(int l=0; l<nextrap; ++l){

            int depthIdx = l*nf*nx*length_M;

            extrapDepth<<<nBlocks, nThreads, 0, streams[is]>>>(&d_new_forw[is*sizePulse], nf, nx, \
                M, &d_w_op_forw[depthIdx], &d_old_forw[is*sizePulse]);
            
            extrapDepth<<<nBlocks, nThreads, 0, streams[is]>>>(&d_new_back[is*sizePulse], nf, nx, \
                M, &d_w_op_back[depthIdx], &d_old_back[is*sizePulse]);
            
            imaging<<<1, nx>>>(&d_image[is*sizeImage + l*nx], &d_new_forw[is*sizePulse], \
                &d_new_back[is*sizePulse], nf, nx, M);
            
            copyPadded<<<nBlocks, nThreads>>>(&d_old_forw[is*sizePulse], &d_new_forw[is*sizePulse],\
                nf, nx, M);

            copyPadded<<<nBlocks, nThreads>>>(&d_old_back[is*sizePulse], &d_new_back[is*sizePulse],\
                nf, nx, M);
            
        }
        hipMemcpyAsync(h_forw_pulses[is].wf, &d_new_forw[is*sizePulse], \
            sizePulse*sizeof(fcomp), hipMemcpyDeviceToHost, streams[is]);
        hipMemcpyAsync(h_back_pulses[is].wf, &d_new_back[is*sizePulse], \
            sizePulse*sizeof(fcomp), hipMemcpyDeviceToHost, streams[is]);
        hipMemcpyAsync(&h_image[is*sizeImage], &d_image[is*sizeImage], \
            sizeImage*sizeof(fcomp), hipMemcpyDeviceToHost, streams[is]);

        hipStreamDestroy(streams[is]);
    }

    startTime = std::chrono::high_resolution_clock::now();
    //copy to unpadded memory
    for(int is=0; is<ns; ++is)
        for (int j=0; j<nf; ++j)
            for (int i=0; i<nx; ++i){
                forw_pulse[is*sizePulse + j*nx + i] = h_forw_pulses[is].wf[j*dim_x + i + M];
                back_pulse[is*sizePulse + j*nx + i] = h_back_pulses[is].wf[j*dim_x + i + M];
            }
    stopTime = std::chrono::high_resolution_clock::now();
    elapsedTime = stopTime - startTime;
    std::cout << "Read wavefileds back time(s): " << elapsedTime.count() / 1000.0 << " seconds." << std::endl;
        
    startTime = std::chrono::high_resolution_clock::now();
    //take real part of images
    for(int is=0; is<ns; ++is)
        for(int l=0; l<nextrap; ++l)
            for(int i=0; i<nx; ++i){
                image[is*sizeImage + l*nx + i] = reinterpret_cast<float*>(h_image)[2*(is*sizeImage + l*nx + i)];
            }
    stopTime = std::chrono::high_resolution_clock::now();
    elapsedTime = stopTime - startTime;
    std::cout << "Read image real part time(s): " << elapsedTime.count() / 1000.0 << " seconds." << std::endl;

    startTime = std::chrono::high_resolution_clock::now();
    //free host memory
    delete [] h_image;
    delete [] h_w_op_forw;
    delete [] h_w_op_back;
    stopTime = std::chrono::high_resolution_clock::now();
    elapsedTime = stopTime - startTime;
    std::cout << "Free host memory time(s): " << elapsedTime.count() / 1000.0 << " seconds." << std::endl;

    //free device memory
    hipFree(d_w_op_forw);
    hipFree(d_w_op_back);
    hipFree(d_new_forw);
    hipFree(d_old_forw);
    hipFree(d_new_back);
    hipFree(d_old_back);
    hipFree(d_image);

}

} //end extern "C"
